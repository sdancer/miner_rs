#include "hip/hip_runtime.h"
typedef unsigned long long uint64_t;
typedef unsigned int  uint32_t;
typedef unsigned char uint8_t;

using u32 = uint32_t;
using u64 = uint64_t;
using u8  = uint8_t;
 
const u32 OUT_LEN = 32;
const u32 KEY_LEN = 32;
const u32 BLOCK_LEN = 64;
const u32 CHUNK_LEN = 1024;
// Multiple chunks make a snicker bar :)
const u32 SNICKER = 1U << 10;
// Factory height and snicker size have an inversly propotional relationship
// FACTORY_HT * (log2 SNICKER) + 10 >= 64 
const u32 FACTORY_HT = 5;

const u32 CHUNK_START = 1 << 0;
const u32 CHUNK_END = 1 << 1;
const u32 PARENT = 1 << 2;
const u32 ROOT = 1 << 3;
const u32 KEYED_HASH = 1 << 4;
const u32 DERIVE_KEY_CONTEXT = 1 << 5;
const u32 DERIVE_KEY_MATERIAL = 1 << 6;

const int usize = sizeof(u32) * 8;

// redefine functions, but for the GPU
// all of them are the same but with g_ prefixed
__constant__ const u32 g_IV[8] = {
    0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 
    0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19,
};

__device__ __forceinline__ uint32_t g_rotr32(uint32_t v, int s) {
    return (v >> s) | (v << (32 - s));
}

#define GG(a,b,c,d, mx,my)         \
    do {                           \
        (a) += (b) + (mx);         \
        (d)  = g_rotr32((d) ^ (a), 16); \
        (c) += (d);                \
        (b)  = g_rotr32((b) ^ (c), 12); \
        (a) += (b) + (my);         \
        (d)  = g_rotr32((d) ^ (a), 8);  \
        (c) += (d);                \
        (b)  = g_rotr32((b) ^ (c), 7);  \
    } while (0)

// m can be passed as separate regs (m0..m15) or you can load them from memory first.
// This keeps *state* entirely in registers via references.
__device__ __forceinline__ void g_round_regs(
    uint32_t &s0,  uint32_t &s1,  uint32_t &s2,  uint32_t &s3,
    uint32_t &s4,  uint32_t &s5,  uint32_t &s6,  uint32_t &s7,
    uint32_t &s8,  uint32_t &s9,  uint32_t &s10, uint32_t &s11,
    uint32_t &s12, uint32_t &s13, uint32_t &s14, uint32_t &s15,
    const uint32_t m0,  const uint32_t m1,  const uint32_t m2,  const uint32_t m3,
    const uint32_t m4,  const uint32_t m5,  const uint32_t m6,  const uint32_t m7,
    const uint32_t m8,  const uint32_t m9,  const uint32_t m10, const uint32_t m11,
    const uint32_t m12, const uint32_t m13, const uint32_t m14, const uint32_t m15)
{
    // Mix the columns.
    GG(s0,  s4,  s8,  s12, m0,  m1);
    GG(s1,  s5,  s9,  s13, m2,  m3);
    GG(s2,  s6,  s10, s14, m4,  m5);
    GG(s3,  s7,  s11, s15, m6,  m7);
    // Mix the diagonals.
    GG(s0,  s5,  s10, s15, m8,  m9);
    GG(s1,  s6,  s11, s12, m10, m11);
    GG(s2,  s7,  s8,  s13, m12, m13);
    GG(s3,  s4,  s9,  s14, m14, m15);
}

__device__ __forceinline__ void g_permute_regs(
    u32 &m0,  u32 &m1,  u32 &m2,  u32 &m3,
    u32 &m4,  u32 &m5,  u32 &m6,  u32 &m7,
    u32 &m8,  u32 &m9,  u32 &m10, u32 &m11,
    u32 &m12, u32 &m13, u32 &m14, u32 &m15)
{
    // BLAKE3 message permutation:
    // [2,6,3,10,7,0,4,13,1,11,12,5,9,14,15,8]
    u32 t0=m0, t1=m1, t2=m2, t3=m3, t4=m4, t5=m5, t6=m6, t7=m7;
    u32 t8=m8, t9=m9, t10=m10, t11=m11, t12=m12, t13=m13, t14=m14, t15=m15;

    m0  = t2;   m1  = t6;   m2  = t3;   m3  = t10;
    m4  = t7;   m5  = t0;   m6  = t4;   m7  = t13;
    m8  = t1;   m9  = t11;  m10 = t12;  m11 = t5;
    m12 = t9;   m13 = t14;  m14 = t15;  m15 = t8;
}

__device__ __forceinline__ void g_compress(
    const u32 *__restrict__ chaining_value,  // cv[8]
    const u32 *__restrict__ block_words,     // m[16]
    u64 counter,
    u32 block_len,
    u32 flags,
    u32 *__restrict__ state_out)             // writes v[16]
{
    // ---- Load CV into regs (keep originals for feedforward) ----
    u32 cv0 = chaining_value[0], cv1 = chaining_value[1];
    u32 cv2 = chaining_value[2], cv3 = chaining_value[3];
    u32 cv4 = chaining_value[4], cv5 = chaining_value[5];
    u32 cv6 = chaining_value[6], cv7 = chaining_value[7];

    // Working state in 16 registers
    u32 s0=cv0, s1=cv1, s2=cv2, s3=cv3, s4=cv4, s5=cv5, s6=cv6, s7=cv7;
    u32 s8=g_IV[0], s9=g_IV[1], s10=g_IV[2], s11=g_IV[3];
    u32 s12=(u32)counter, s13=(u32)(counter >> 32), s14=block_len, s15=flags;

    // ---- Load message into regs ----
    u32 m0 = block_words[0],  m1  = block_words[1];
    u32 m2 = block_words[2],  m3  = block_words[3];
    u32 m4 = block_words[4],  m5  = block_words[5];
    u32 m6 = block_words[6],  m7  = block_words[7];
    u32 m8 = block_words[8],  m9  = block_words[9];
    u32 m10= block_words[10], m11 = block_words[11];
    u32 m12= block_words[12], m13 = block_words[13];
    u32 m14= block_words[14], m15 = block_words[15];

    // Round 0: identity
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m0,m1,m2,m3,m4,m5,m6,m7,m8,m9,m10,m11,m12,m13,m14,m15);
    
    // Round 1: perm^1 = [2,6,3,10,7,0,4,13,1,11,12,5,9,14,15,8]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m2,m6,m3,m10,m7,m0,m4,m13,m1,m11,m12,m5,m9,m14,m15,m8);
    
    // Round 2: perm^2 = [3,4,10,12,13,2,7,14,6,5,9,0,11,15,8,1]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m3,m4,m10,m12,m13,m2,m7,m14,m6,m5,m9,m0,m11,m15,m8,m1);
    
    // Round 3: perm^3 = [10,7,12,9,14,3,13,15,4,0,11,2,5,8,1,6]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m10,m7,m12,m9,m14,m3,m13,m15,m4,m0,m11,m2,m5,m8,m1,m6);
    
    // Round 4: perm^4 = [12,13,9,11,15,10,14,8,7,2,5,3,0,1,6,4]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m12,m13,m9,m11,m15,m10,m14,m8,m7,m2,m5,m3,m0,m1,m6,m4);
    
    // Round 5: perm^5 = [9,14,11,5,8,12,15,1,13,3,0,10,2,6,4,7]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m9,m14,m11,m5,m8,m12,m15,m1,m13,m3,m0,m10,m2,m6,m4,m7);
    
    // Round 6: perm^6 = [11,15,5,0,1,9,8,6,14,10,2,12,3,4,7,13]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m11,m15,m5,m0,m1,m9,m8,m6,m14,m10,m2,m12,m3,m4,m7,m13);

    // ---- Feedforward / output transform ----
    s0  ^= s8;   s1  ^= s9;   s2  ^= s10;  s3  ^= s11;
    s4  ^= s12;  s5  ^= s13;  s6  ^= s14;  s7  ^= s15;

    s8  ^= cv0;  s9  ^= cv1;  s10 ^= cv2;  s11 ^= cv3;
    s12 ^= cv4;  s13 ^= cv5;  s14 ^= cv6;  s15 ^= cv7;

    // ---- Write out ----
    state_out[0]=s0;  state_out[1]=s1;   state_out[2]=s2;   state_out[3]=s3;
    state_out[4]=s4;  state_out[5]=s5;   state_out[6]=s6;   state_out[7]=s7;
    state_out[8]=s8;  state_out[9]=s9;   state_out[10]=s10; state_out[11]=s11;
    state_out[12]=s12;state_out[13]=s13; state_out[14]=s14; state_out[15]=s15;
}


extern "C" __global__ void compress(
    u32 *__restrict__ chaining_value,  // cv[8]
    u32 *__restrict__ block_words,     // m[16]
    u64 counter,
    u32 block_len,
    u32 flags,
    u32 *__restrict__ state_out)             // writes v[16]
{
 
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW == 0 && COL == 0) {
      g_compress(chaining_value, block_words, counter & 0xffffffff, block_len, flags, state_out);
      printf("got called %lx\n",state_out[0]);
    }
}

