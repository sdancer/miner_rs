#include "hip/hip_runtime.h"
#include <sm_61_intrinsics.h>

typedef unsigned long long uint64_t;
typedef unsigned int  uint32_t;
typedef unsigned char uint8_t;

typedef signed int  int32_t;
typedef signed char int8_t;

using u32 = uint32_t;
using u64 = uint64_t;
using u8  = uint8_t;
 

//const u32 OUT_LEN = 32;
//const u32 KEY_LEN = 32;
//const u32 BLOCK_LEN = 64;
//const u32 CHUNK_LEN = 1024;
//// Multiple chunks make a snicker bar :)
//const u32 SNICKER = 1U << 10;
//// Factory height and snicker size have an inversly propotional relationship
//// FACTORY_HT * (log2 SNICKER) + 10 >= 64 
//const u32 FACTORY_HT = 5;
//
const u32 CHUNK_START = 1 << 0;
const u32 CHUNK_END = 1 << 1;
//const u32 PARENT = 1 << 2;
const u32 ROOT = 1 << 3;
//const u32 KEYED_HASH = 1 << 4;
//const u32 DERIVE_KEY_CONTEXT = 1 << 5;
//const u32 DERIVE_KEY_MATERIAL = 1 << 6;

//const int usize = sizeof(u32) * 8;

// redefine functions, but for the GPU
// all of them are the same but with g_ prefixed
__constant__ const u32 g_IV[8] = {
    0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 
    0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19,
};


#ifdef NEWGCOMP
__device__ __forceinline__ uint32_t g_rotr32(uint32_t v, int s) {
    return (v >> s) | (v << (32 - s));
}

#define GG(a,b,c,d, mx,my)         \
    do {                           \
        (a) += (b) + (mx);         \
        (d)  = g_rotr32((d) ^ (a), 16); \
        (c) += (d);                \
        (b)  = g_rotr32((b) ^ (c), 12); \
        (a) += (b) + (my);         \
        (d)  = g_rotr32((d) ^ (a), 8);  \
        (c) += (d);                \
        (b)  = g_rotr32((b) ^ (c), 7);  \
    } while (0)

// m can be passed as separate regs (m0..m15) or you can load them from memory first.
// This keeps *state* entirely in registers via references.
__device__ __forceinline__ void g_round_regs(
    uint32_t &s0,  uint32_t &s1,  uint32_t &s2,  uint32_t &s3,
    uint32_t &s4,  uint32_t &s5,  uint32_t &s6,  uint32_t &s7,
    uint32_t &s8,  uint32_t &s9,  uint32_t &s10, uint32_t &s11,
    uint32_t &s12, uint32_t &s13, uint32_t &s14, uint32_t &s15,
    const uint32_t m0,  const uint32_t m1,  const uint32_t m2,  const uint32_t m3,
    const uint32_t m4,  const uint32_t m5,  const uint32_t m6,  const uint32_t m7,
    const uint32_t m8,  const uint32_t m9,  const uint32_t m10, const uint32_t m11,
    const uint32_t m12, const uint32_t m13, const uint32_t m14, const uint32_t m15)
{
    // Mix the columns.
    GG(s0,  s4,  s8,  s12, m0,  m1);
    GG(s1,  s5,  s9,  s13, m2,  m3);
    GG(s2,  s6,  s10, s14, m4,  m5);
    GG(s3,  s7,  s11, s15, m6,  m7);
    // Mix the diagonals.
    GG(s0,  s5,  s10, s15, m8,  m9);
    GG(s1,  s6,  s11, s12, m10, m11);
    GG(s2,  s7,  s8,  s13, m12, m13);
    GG(s3,  s4,  s9,  s14, m14, m15);
}

__device__ __forceinline__ void g_permute_regs(
    u32 &m0,  u32 &m1,  u32 &m2,  u32 &m3,
    u32 &m4,  u32 &m5,  u32 &m6,  u32 &m7,
    u32 &m8,  u32 &m9,  u32 &m10, u32 &m11,
    u32 &m12, u32 &m13, u32 &m14, u32 &m15)
{
    // BLAKE3 message permutation:
    // [2,6,3,10,7,0,4,13,1,11,12,5,9,14,15,8]
    u32 t0=m0, t1=m1, t2=m2, t3=m3, t4=m4, t5=m5, t6=m6, t7=m7;
    u32 t8=m8, t9=m9, t10=m10, t11=m11, t12=m12, t13=m13, t14=m14, t15=m15;

    m0  = t2;   m1  = t6;   m2  = t3;   m3  = t10;
    m4  = t7;   m5  = t0;   m6  = t4;   m7  = t13;
    m8  = t1;   m9  = t11;  m10 = t12;  m11 = t5;
    m12 = t9;   m13 = t14;  m14 = t15;  m15 = t8;
}

__device__ __forceinline__ void g_compress(
    const u32 *__restrict__ chaining_value,  // cv[8]
    const u32 *__restrict__ block_words,     // m[16]
    u32 counter,
    u32 block_len,
    u32 flags,
    u32 *__restrict__ state_out)             // writes v[16]
{
    // ---- Load CV into regs (keep originals for feedforward) ----
    u32 cv0 = chaining_value[0], cv1 = chaining_value[1];
    u32 cv2 = chaining_value[2], cv3 = chaining_value[3];
    u32 cv4 = chaining_value[4], cv5 = chaining_value[5];
    u32 cv6 = chaining_value[6], cv7 = chaining_value[7];

    // Working state in 16 registers
    u32 s0=cv0, s1=cv1, s2=cv2, s3=cv3, s4=cv4, s5=cv5, s6=cv6, s7=cv7;
    u32 s8=g_IV[0], s9=g_IV[1], s10=g_IV[2], s11=g_IV[3];
    u32 s12=(u32)counter, s13=0, s14=block_len, s15=flags;

    // ---- Load message into regs ----
    u32 m0 = block_words[0],  m1  = block_words[1];
    u32 m2 = block_words[2],  m3  = block_words[3];
    u32 m4 = block_words[4],  m5  = block_words[5];
    u32 m6 = block_words[6],  m7  = block_words[7];
    u32 m8 = block_words[8],  m9  = block_words[9];
    u32 m10= block_words[10], m11 = block_words[11];
    u32 m12= block_words[12], m13 = block_words[13];
    u32 m14= block_words[14], m15 = block_words[15];

    // Round 0: identity
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m0,m1,m2,m3,m4,m5,m6,m7,m8,m9,m10,m11,m12,m13,m14,m15);
    
    // Round 1: perm^1 = [2,6,3,10,7,0,4,13,1,11,12,5,9,14,15,8]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m2,m6,m3,m10,m7,m0,m4,m13,m1,m11,m12,m5,m9,m14,m15,m8);
    
    // Round 2: perm^2 = [3,4,10,12,13,2,7,14,6,5,9,0,11,15,8,1]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m3,m4,m10,m12,m13,m2,m7,m14,m6,m5,m9,m0,m11,m15,m8,m1);
    
    // Round 3: perm^3 = [10,7,12,9,14,3,13,15,4,0,11,2,5,8,1,6]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m10,m7,m12,m9,m14,m3,m13,m15,m4,m0,m11,m2,m5,m8,m1,m6);
    
    // Round 4: perm^4 = [12,13,9,11,15,10,14,8,7,2,5,3,0,1,6,4]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m12,m13,m9,m11,m15,m10,m14,m8,m7,m2,m5,m3,m0,m1,m6,m4);
    
    // Round 5: perm^5 = [9,14,11,5,8,12,15,1,13,3,0,10,2,6,4,7]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m9,m14,m11,m5,m8,m12,m15,m1,m13,m3,m0,m10,m2,m6,m4,m7);
    
    // Round 6: perm^6 = [11,15,5,0,1,9,8,6,14,10,2,12,3,4,7,13]
    g_round_regs(s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15,
                 m11,m15,m5,m0,m1,m9,m8,m6,m14,m10,m2,m12,m3,m4,m7,m13);

    // ---- Feedforward / output transform ----
    s0  ^= s8;   s1  ^= s9;   s2  ^= s10;  s3  ^= s11;
    s4  ^= s12;  s5  ^= s13;  s6  ^= s14;  s7  ^= s15;

    s8  ^= cv0;  s9  ^= cv1;  s10 ^= cv2;  s11 ^= cv3;
    s12 ^= cv4;  s13 ^= cv5;  s14 ^= cv6;  s15 ^= cv7;

    // ---- Write out ----
    state_out[0]=s0;  state_out[1]=s1;   state_out[2]=s2;   state_out[3]=s3;
    state_out[4]=s4;  state_out[5]=s5;   state_out[6]=s6;   state_out[7]=s7;
    state_out[8]=s8;  state_out[9]=s9;   state_out[10]=s10; state_out[11]=s11;
    state_out[12]=s12;state_out[13]=s13; state_out[14]=s14; state_out[15]=s15;
}
#else
__constant__ const int g_MSG_PERMUTATION[] = {
    2, 6, 3, 10, 7, 0, 4, 13,
    1, 11, 12, 5, 9, 14, 15, 8
};

__device__ __forceinline__ u32 g_rotr(u32 value, int shift) {
    // OPTIMIZATION: Use fast bit rotation with compiler intrinsics for lower precision/higher speed
    return __funnelshift_r(value, value, shift);
}

__device__ __forceinline__ void g_g(u32 state[16], u32 a, u32 b, u32 c, u32 d, u32 mx, u32 my) {
    // OPTIMIZATION: Fast arithmetic with reduced precision (CPU verification will catch errors)
    // Use fast unchecked arithmetic - overflow is acceptable for speed
    u32 temp_a = state[a] + state[b] + mx; // Fast unchecked addition
    state[d] = g_rotr((state[d] ^ temp_a), 16);
    u32 temp_c = state[c] + state[d];

    state[b] = g_rotr((state[b] ^ temp_c), 12);
    temp_a += state[b] + my; // Fast unchecked addition
    state[d] = g_rotr((state[d] ^ temp_a), 8);

    temp_c += state[d]; // Fast unchecked addition
    state[b] = g_rotr((state[b] ^ temp_c), 7);

    // Write back results
    state[a] = temp_a;
    state[c] = temp_c;
}

__device__ void g_round(u32 state[16], u32 m[16]) {
    // Mix the columns.
    g_g(state, 0, 4, 8, 12, m[0], m[1]);
    g_g(state, 1, 5, 9, 13, m[2], m[3]);
    g_g(state, 2, 6, 10, 14, m[4], m[5]);
    g_g(state, 3, 7, 11, 15, m[6], m[7]);
    // Mix the diagonals.
    g_g(state, 0, 5, 10, 15, m[8], m[9]);
    g_g(state, 1, 6, 11, 12, m[10], m[11]);
    g_g(state, 2, 7, 8, 13, m[12], m[13]);
    g_g(state, 3, 4, 9, 14, m[14], m[15]);
}

__device__ void g_permute(u32 m[16]) {
    u32 permuted[16];
    for(int i=0; i<16; i++)
        permuted[i] = m[g_MSG_PERMUTATION[i]];
    for(int i=0; i<16; i++)
        m[i] = permuted[i];
}

// custom memcpy, apparently cuda's memcpy is slow
// when called within a kernel
__device__ void g_memcpy(u32 *lhs, const u32 *rhs, int size) {
    // assuming u32 is 4 bytes
    int len = size / 4;
    for(int i=0; i<len; i++)
        lhs[i] = rhs[i];
}

// custom memset
template<typename T, typename ptr_t>
__device__ void g_memset(ptr_t dest, T val, int count) {
    for(int i=0; i<count; i++)
        dest[i] = val;
}

__device__ __forceinline__ void g_compress(
    const u32 *chaining_value,
    const u32 *block_words,
    u64 counter,
    u32 block_len,
    u32 flags,
    u32 *state
) {

    g_memcpy(state, chaining_value, 32);
    g_memcpy(state+8, g_IV, 16);
    state[12] = (u32)counter;
    state[13] = (u32)(counter >> 32);
    state[14] = block_len;
    state[15] = flags;

    u32 block[16];
    g_memcpy(block, block_words, 64);

    g_round(state, block); // round 1

    g_permute(block);
    g_round(state, block); // round 2

    g_permute(block);
    g_round(state, block); // round 3

    g_permute(block);
    g_round(state, block); // round 4

    g_permute(block);
    g_round(state, block); // round 5

    g_permute(block);
    g_round(state, block); // round 6

    g_permute(block);
    g_round(state, block); // round 7


    for(int i = 0; i < 8; i++){
        state[i] ^= state[i + 8];
    }
}
#endif


extern "C" __global__ void compress(
    u32 *__restrict__ chaining_value,  // cv[8]
    u32 *__restrict__ block_words,     // m[16]
    u64 counter,
    u32 block_len,
    u32 flags,
    u32 *__restrict__ state_out)             // writes v[16]
{
 
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW == 0 && COL == 0) {
      g_compress(chaining_value, block_words, counter & 0xffffffff, block_len, flags, state_out);
 //     printf("got called %lx\n",state_out[0]);
    }
}


// --- Emit one 64B XOF block into 16 u32 words (dstw)
__device__ inline void xof_emit_words(
    u32 blk,
    const u32 root[8],
    const u32 precv[8],
    const u32 last_words[16],
    u32 last_len,
    u32 dstw[16])
{
    u32 out[16];

    const uint64_t t = (uint64_t)blk;
    //printf("blk: %x\n", blk);

    const u32 flags = (CHUNK_END | ROOT);
    // g_compress writes 16 words to state; its low half is already lo^hi (root CV),
    // high half is the raw hi (no feed-forward).
    g_compress(precv, const_cast<u32*>(last_words), t, last_len, flags, out);

    #pragma unroll
    for (int w=0; w<8; ++w) dstw[w] = out[w];

    #pragma unroll
    for (int w=0; w<8; ++w) dstw[8+w] = out[8+w] ^ precv[w];
}

#ifndef TILE_K
#define TILE_K 256  // multiple of 64 and 4
#endif


// Helpers
__device__ __forceinline__ void store_le64(uint8_t* dst, u64 x) {
    #pragma unroll
    for (int i = 0; i < 8; ++i) dst[i] = (uint8_t)(x >> (8*i));
}

// Compute root + pre-final CV + last words from a 240B seed
__device__ __forceinline__
void compute_root_from_seed240(const uint8_t* __restrict__ seed240,
                               u32* __restrict__ out_root,   // 8
                               u32* __restrict__ out_precv,  // 8
                               u32* __restrict__ out_lwords, // 16
                               uint8_t* __restrict__ out_llen)
{
    u32 cv[8];
    #pragma unroll
    for (int i = 0; i < 8; ++i) cv[i] = g_IV[i];

    u32 m[16], st[16];

    for (int blk = 0; blk < 4; ++blk) {
        const u32 blen = (blk == 3 ? 48u : 64u);
        // pack 64B (or 48B final) into 16 u32 (zero padded)
        #pragma unroll
        for (int w = 0; w < 16; ++w) m[w] = 0u;
        #pragma unroll
        for (u32 i = 0; i < blen; ++i)
            reinterpret_cast<uint8_t*>(m)[i] = seed240[blk*64 + i];

        u32 flags = 0;
        if (blk == 0) flags |= CHUNK_START;
        if (blk == 3) flags |= (CHUNK_END | ROOT);

        if (blk == 3) {
            #pragma unroll
            for (int w = 0; w < 8;  ++w) out_precv[w]  = cv[w];
            #pragma unroll
            for (int w = 0; w < 16; ++w) out_lwords[w] = m[w];
            *out_llen = (uint8_t)blen; // 48
        }

        g_compress(cv, m, 0ULL, blen, flags, st);
        #pragma unroll
        for (int w = 0; w < 8; ++w) cv[w] = st[w];
    }

    #pragma unroll
    for (int w = 0; w < 8; ++w) out_root[w] = cv[w];
}

// -----------------------------------------------------------------------------
// One kernel does everything for a range of nonces.
// Grid:  grid.x = #seeds (or any >=1, kernel loops by stride), block = (16,16)
// Smem:  dynamic = (16*TILE_K + TILE_K*16) bytes
// Args:
//   d_prefix232 : same 232B prefix for all seeds
//   nonce_start : starting 64-bit nonce (little-endian written into bytes 232..239)
//   nonce_count : number of seeds (one 16x16 output per nonce)
//   d_C         : outputs [nonce_count][16][16] int32
// -----------------------------------------------------------------------------
#ifndef TILE_K
#define TILE_K 256
#endif


extern "C" __global__
__launch_bounds__(256, 4)
void solve_nonce_range_fused(
        const uint8_t* __restrict__ d_prefix232, // 232 bytes
        unsigned long long* d_iter_count,
        u64 nonce_start,
        int nonce_count,
        u32* __restrict__ d_hashes /* (still unused; hashing kept commented) */)
{
    const int i = threadIdx.y;   // 0..15
    const int j = threadIdx.x;   // 0..15

    // ---- Static small shared (persists across tile iters) ----
    __shared__ uint8_t sh_prefix[232];
    __shared__ uint8_t sh_seed[240];
    __shared__ u32 sh_root[8];
    __shared__ u32 sh_precv[8];
    __shared__ u32 sh_lwords[16];
    __shared__ uint8_t sh_llen; // = 48

    // Each seed’s 16x16 accumulators (1024B) to be hashed on-chip (kept for later)
    __shared__ int32_t tileC[16 * 16];

    // Copy the common 232B prefix once per block
    if (i == 0 && j == 0) {
        #pragma unroll
        for (int t = 0; t < 232; ++t) sh_prefix[t] = d_prefix232[t];
    }
    __syncthreads();

    // ---- Dynamic shared for tiles (matches your matmul kernel) ----
    extern __shared__ __align__(16) uint8_t smem[];
    uint8_t* As = smem;                                  // 16 * TILE_K bytes
    uint8_t* Bs = smem + (size_t)16 * TILE_K + 32;       // + small pad to reduce bank conflicts

    // Process many nonces with a single persistent block (optional, good for large ranges)
    for (int seed = blockIdx.x; seed < nonce_count; seed += gridDim.x) {

        // Thread (0,0) builds the 240B seed and computes root/preCV/lastWords
        if (i == 0 && j == 0) {
            atomicAdd(d_iter_count, 1ULL);

            // prefix[0..231]
            #pragma unroll
            for (int t = 0; t < 232; ++t) sh_seed[t] = sh_prefix[t];

            // nonce (LE) into bytes 232..239
            const u64 nonce = nonce_start + (u64)seed;
            store_le64(&sh_seed[232], nonce);

            // derive root/preCV/lastWords/lastLen
            compute_root_from_seed240(sh_seed, sh_root, sh_precv, sh_lwords, &sh_llen);
        }
        __syncthreads();

        // --- Matmul 16xK by Kx16 with on-the-fly XOF using sh_root/sh_precv/sh_lwords ---
        constexpr int K            = 50240;
        constexpr int A_BYTES      = 16 * K;           // 803,840
        constexpr int A_BLOCKS     = A_BYTES / 64;     // 12,560
        constexpr int B_BASE_BLOCK = A_BLOCKS;         // 12,560
        const u32     llen         = (u32)sh_llen;

        int acc   = 0;
        int sum_b = 0;

        // Flattened thread id for coalesced cooperative loads
        const int thread_id     = threadIdx.y * blockDim.x + threadIdx.x; // 0..255
        const int total_threads = blockDim.x * blockDim.y;                 // 256

        for (int k0 = 0; k0 < K; k0 += TILE_K) {
            const int tile = min(TILE_K, K - k0);

            // ---- Produce A tile into As (by rows), coalesced via linear thread id ----
            const int a_blocks_per_row = (tile + 63) / 64;
            for (int linear_idx = thread_id; linear_idx < 16 * a_blocks_per_row; linear_idx += total_threads) {
                const int ri      = linear_idx / a_blocks_per_row;  // row 0..15
                const int rb      = linear_idx % a_blocks_per_row;  // 64-B block index within the row
                const int kk_base = rb * 64;

                if (kk_base < tile) {
                    const uint32_t blkA = (uint32_t)(ri * (K/64) + (k0/64) + rb);

                    u32 words[16];
                    xof_emit_words(blkA, sh_root, sh_precv, sh_lwords, llen, words);

                    // Vectorized store: 64B = 4×uint4 (16B each)
                    uint8_t* dst_byte = As + (size_t)ri * TILE_K + kk_base;
                    uint4*   dst_vec  = reinterpret_cast<uint4*>(dst_byte);
                    uint4*   src_vec  = reinterpret_cast<uint4*>(words);

                    #pragma unroll
                    for (int v = 0; v < 4; ++v) {
                        const int byte_off = v * 16;
                        if (kk_base + byte_off < tile) {
                            dst_vec[v] = src_vec[v];
                        }
                    }
                }
            }

            // ---- Produce B tile into Bs (by columns), vectorized scatter ----
            const int b_blocks = (tile + 3) / 4; // 4 bytes per col-chunk
            for (int gb = thread_id; gb < b_blocks; gb += total_threads) {
                const int kk_base = gb * 4;
                const uint32_t blkB = (uint32_t)(B_BASE_BLOCK + ((k0 + kk_base) >> 2));

                u32 words[16];
                xof_emit_words(blkB, sh_root, sh_precv, sh_lwords, llen, words);

                // For each of 4 columns in this 64-B block, drop one uint4 (16B) into Bs
                const uint4* srcw_vec = reinterpret_cast<const uint4*>(words);
                #pragma unroll
                for (int q = 0; q < 4; ++q) {
                    const int kk = kk_base + q;
                    if (kk < tile) {
                        uint4* dstw_vec = reinterpret_cast<uint4*>(Bs + (size_t)kk * 16);
                        *dstw_vec = srcw_vec[q];
                    }
                }
            }
            __syncthreads();

            // ---- DP4A accumulate with aggressive unrolling ----
            int kk = 0;

            // 16-wide block (4×DP4A) for maximal ILP
            for (; kk + 15 < tile; kk += 16) {
                // Load 4×uint32 (16 bytes) of A as four 4-tuples
                uint32_t a_vec1 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 0);
                uint32_t a_vec2 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 4);
                uint32_t a_vec3 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 8);
                uint32_t a_vec4 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 12);

                auto pack4 = [](uint32_t r)->int {
                    int a0 = (int)((r >>  0) & 0xFF) - 128;
                    int a1 = (int)((r >>  8) & 0xFF) - 128;
                    int a2 = (int)((r >> 16) & 0xFF) - 128;
                    int a3 = (int)((r >> 24) & 0xFF) - 128;
                    return (a0 & 0xFF) | ((a1 & 0xFF) << 8) | ((a2 & 0xFF) << 16) | ((a3 & 0xFF) << 24);
                };

                int a_p1 = pack4(a_vec1);
                int a_p2 = pack4(a_vec2);
                int a_p3 = pack4(a_vec3);
                int a_p4 = pack4(a_vec4);

                // Load B as 16 separate int8, then pack in 4-tuples
                #define B_AT(off) ((int)((int8_t)Bs[(size_t)(kk + (off)) * 16 + j]))
                int b0=B_AT(0),  b1=B_AT(1),  b2=B_AT(2),  b3=B_AT(3);
                int b4=B_AT(4),  b5=B_AT(5),  b6=B_AT(6),  b7=B_AT(7);
                int b8=B_AT(8),  b9=B_AT(9),  b10=B_AT(10), b11=B_AT(11);
                int b12=B_AT(12),b13=B_AT(13),b14=B_AT(14), b15=B_AT(15);
                #undef B_AT

                int b_p1 = (b0 & 0xFF) | ((b1 & 0xFF) << 8) | ((b2 & 0xFF) << 16) | ((b3 & 0xFF) << 24);
                int b_p2 = (b4 & 0xFF) | ((b5 & 0xFF) << 8) | ((b6 & 0xFF) << 16) | ((b7 & 0xFF) << 24);
                int b_p3 = (b8 & 0xFF) | ((b9 & 0xFF) << 8) | ((b10 & 0xFF) << 16) | ((b11 & 0xFF) << 24);
                int b_p4 = (b12 & 0xFF) | ((b13 & 0xFF) << 8) | ((b14 & 0xFF) << 16) | ((b15 & 0xFF) << 24);

                sum_b += (b0 + b1 + b2 + b3 +
                          b4 + b5 + b6 + b7 +
                          b8 + b9 + b10 + b11 +
                          b12 + b13 + b14 + b15);

                acc = __dp4a(a_p1, b_p1, acc);
                acc = __dp4a(a_p2, b_p2, acc);
                acc = __dp4a(a_p3, b_p3, acc);
                acc = __dp4a(a_p4, b_p4, acc);
            }

            // 8-wide block (2×DP4A)
            for (; kk + 7 < tile; kk += 8) {
                uint32_t a_vec1 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 0);
                uint32_t a_vec2 = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk + 4);

                auto pack4s = [](uint32_t r)->int {
                    int a0 = (int)((r >>  0) & 0xFF) - 128;
                    int a1 = (int)((r >>  8) & 0xFF) - 128;
                    int a2 = (int)((r >> 16) & 0xFF) - 128;
                    int a3 = (int)((r >> 24) & 0xFF) - 128;
                    return (a0 & 0xFF) | ((a1 & 0xFF) << 8) | ((a2 & 0xFF) << 16) | ((a3 & 0xFF) << 24);
                };

                int a_p1 = pack4s(a_vec1);
                int a_p2 = pack4s(a_vec2);

                int b0 = (int)((int8_t)Bs[(size_t)(kk + 0) * 16 + j]);
                int b1 = (int)((int8_t)Bs[(size_t)(kk + 1) * 16 + j]);
                int b2 = (int)((int8_t)Bs[(size_t)(kk + 2) * 16 + j]);
                int b3 = (int)((int8_t)Bs[(size_t)(kk + 3) * 16 + j]);
                int b4 = (int)((int8_t)Bs[(size_t)(kk + 4) * 16 + j]);
                int b5 = (int)((int8_t)Bs[(size_t)(kk + 5) * 16 + j]);
                int b6 = (int)((int8_t)Bs[(size_t)(kk + 6) * 16 + j]);
                int b7 = (int)((int8_t)Bs[(size_t)(kk + 7) * 16 + j]);

                int b_p1 = (b0 & 0xFF) | ((b1 & 0xFF) << 8) | ((b2 & 0xFF) << 16) | ((b3 & 0xFF) << 24);
                int b_p2 = (b4 & 0xFF) | ((b5 & 0xFF) << 8) | ((b6 & 0xFF) << 16) | ((b7 & 0xFF) << 24);

                sum_b += (b0 + b1 + b2 + b3 + b4 + b5 + b6 + b7);
                acc = __dp4a(a_p1, b_p1, acc);
                acc = __dp4a(a_p2, b_p2, acc);
            }

            // 4-wide block (1×DP4A)
            for (; kk + 3 < tile; kk += 4) {
                uint32_t a_vec = *reinterpret_cast<const uint32_t*>(As + (size_t)i*TILE_K + kk);

                int a0 = (int)((a_vec >>  0) & 0xFF) - 128;
                int a1 = (int)((a_vec >>  8) & 0xFF) - 128;
                int a2 = (int)((a_vec >> 16) & 0xFF) - 128;
                int a3 = (int)((a_vec >> 24) & 0xFF) - 128;
                int a_p = (a0 & 0xFF) | ((a1 & 0xFF) << 8) | ((a2 & 0xFF) << 16) | ((a3 & 0xFF) << 24);

                int b0 = (int)((int8_t)Bs[(size_t)(kk + 0) * 16 + j]);
                int b1 = (int)((int8_t)Bs[(size_t)(kk + 1) * 16 + j]);
                int b2 = (int)((int8_t)Bs[(size_t)(kk + 2) * 16 + j]);
                int b3 = (int)((int8_t)Bs[(size_t)(kk + 3) * 16 + j]);
                int b_p = (b0 & 0xFF) | ((b1 & 0xFF) << 8) | ((b2 & 0xFF) << 16) | ((b3 & 0xFF) << 24);

                sum_b += (b0 + b1 + b2 + b3);
                acc = __dp4a(a_p, b_p, acc);
            }

            // Scalar tail
            for (; kk < tile; ++kk) {
                int a_s = (int)((unsigned)As[(size_t)i*TILE_K + kk]) - 128;
                int b_s = (int)((int8_t)Bs[(size_t)kk * 16 + j]);
                acc   += a_s * b_s;
                sum_b += b_s;
            }

            __syncthreads();
        }

        // Un-bias A via shift (128 * sum_b)
        acc += (sum_b << 7);

        // Keep your 16×16 tile in shared (useful if/when you re-enable on-chip hashing)
        tileC[i * 16 + j] = acc;

        // (Hashing path kept commented; re-enable when needed)
        // if (i == 0 && j == 0) { ... g_compress over 16×64B of tileC ... }
        __syncthreads();
        if (i == 0 && j == 0 && seed == 0) { 
            d_hashes[0] = tileC[0];
        }
    }
}


